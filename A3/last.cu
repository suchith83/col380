#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include "modify.cuh"

using namespace std;

#define BLOCK_SIZE 1024

__global__ void compute_frequency(int *matrix, int *freq, int total_size, int max_range) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        int value = matrix[idx];
        if (value >= 0 && value <= max_range) {
            atomicAdd(&freq[value], 1);
        }
    }
}

// scan part of code 


#define BLOCK_SIZE 1024  // Maximum threads per block

__global__ void block_prefix_sum(int *d_input, int *d_output, int *d_block_sums, int n) {
    __shared__ int temp[BLOCK_SIZE];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory
    if (index < n) {
        temp[tid] = d_input[index];
    } else {
        temp[tid] = 0;  // Handle out-of-bounds case
    }
    __syncthreads();

    // **Work-efficient Parallel Scan (Blelloch's Algorithm)**
    // Up-Sweep (Reduction)
    for (int stride = 1; stride < BLOCK_SIZE; stride *= 2) {
        int idx = (tid + 1) * stride * 2 - 1;
        if (idx < BLOCK_SIZE) {
            temp[idx] += temp[idx - stride];
        }
        __syncthreads();
    }

    // Store block sum before zeroing the last element
    if (tid == BLOCK_SIZE - 1) {
        d_block_sums[blockIdx.x] = temp[tid];
        temp[tid] = 0;
    }
    __syncthreads();

    // Down-Sweep (Exclusive Scan)
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
        int idx = (tid + 1) * stride * 2 - 1;
        if (idx < BLOCK_SIZE) {
            int tempVal = temp[idx - stride];
            temp[idx - stride] = temp[idx];
            temp[idx] += tempVal;
        }
        __syncthreads();
    }

    // Write results to global memory
    if (index < n) {
        d_output[index] = temp[tid];
    }
}

__global__ void add_block_sums(int *d_output, int *d_block_sums, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (blockIdx.x > 0 && index < n) {
        d_output[index] += d_block_sums[blockIdx.x - 1];
    }
}

void parallel_prefix_sum(int *d_input, int *d_pref_output, int n) {
    int num_blocks = (n + BLOCK_SIZE - 1) / (BLOCK_SIZE);
    int *d_block_sums;
    hipMalloc(&d_block_sums, num_blocks * sizeof(int));

    block_prefix_sum<<<num_blocks, BLOCK_SIZE>>>(d_input, d_pref_output, d_block_sums, n);
    hipDeviceSynchronize();

    int *h_block_sums = new int[num_blocks];
    hipMemcpy(h_block_sums, d_block_sums, num_blocks * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 1; i < num_blocks; i++) {
        h_block_sums[i] += h_block_sums[i-1];
    }

    hipMemcpy(d_block_sums, h_block_sums, num_blocks * sizeof(int), hipMemcpyHostToDevice);

    add_block_sums<<<num_blocks, BLOCK_SIZE>>>(d_pref_output, d_block_sums, n);
    hipDeviceSynchronize();

    hipFree(d_block_sums);
    delete[] h_block_sums;
    
}

__global__ void modify_matrix(int *matrix, int *prefix_sum, int *output, int total_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        int val = matrix[idx];
        // if (val > 100000000) printf("GPU thread no: %d \n", idx);
        int new_pos = atomicSub(&prefix_sum[val+1], 1) - 1;
        output[new_pos] = val;
    }
}



// with print
vector<vector<vector<int>>> modify(vector<vector<vector<int>>> &matrices, vector<int> &ranges) {
    int N = matrices.size();
    vector<int*> d_matrices(N), d_freqs(N), d_prefix_copy(N), d_output(N);
    // int *h_prefix[N], *h_freq[N];

    for (int i = 0; i < N; i++) {
        int rows = matrices[i].size();
        int cols = matrices[i][0].size();
        int size = rows * cols * sizeof(int);

        vector<int> flat_matrix(rows * cols);
        for (int r = 0; r < rows; r++)
            for (int c = 0; c < cols; c++)
                flat_matrix[r * cols + c] = matrices[i][r][c];

        hipMalloc(&d_matrices[i], size);
        hipMalloc(&d_freqs[i], (ranges[i] + 2) * sizeof(int));
        hipMalloc(&d_prefix_copy[i], (ranges[i] + 2) * sizeof(int));
        hipMalloc(&d_output[i], size);

        hipMemcpy(d_matrices[i], flat_matrix.data(), size, hipMemcpyHostToDevice);
        hipMemset(d_freqs[i], 0, (ranges[i] + 2) * sizeof(int));

        // h_prefix[i] = new int[ranges[i] + 2]();
        // h_freq[i] = new int[ranges[i] + 2]();

        int tot_threads = rows * cols;
        int block_size = 256;
        int noBlocks = (tot_threads + block_size - 1) / block_size;

        compute_frequency<<<noBlocks, block_size>>>(d_matrices[i], d_freqs[i], rows * cols, ranges[i]);
        hipDeviceSynchronize();
        // hipMemcpy(h_freq[i], d_freqs[i], (ranges[i] + 2) * sizeof(int), hipMemcpyDeviceToHost);

        // Print a small range of frequency values
        // printf("Frequency array (first 10 values) for matrix %d:\n", i);
        // for (int j = 54989849 - 5; j < 54989849 + 5; j++) {
        //     printf("%d ", h_freq[i][j]);
        // }
        // printf("\n");
        // cout << ranges[i] << endl;
        // for (int j = 155051280 - 5; j < 155051280 + 5; j++) {
        //     printf("%d ", h_freq[i][j]);
        // }
        // printf("\n");

        parallel_prefix_sum(d_freqs[i], d_prefix_copy[i], ranges[i] + 2);
        // hipMemcpy(h_prefix[i], d_prefix_copy[i], (ranges[i] + 2) * sizeof(int), hipMemcpyDeviceToHost);
        // hipMemcpy(d_prefix_copy[i], h_prefix[i], (ranges[i] + 2) * sizeof(int), hipMemcpyHostToDevice);

        // Print a small range of prefix sum values
        // printf("Prefix sum array (first 10 values) for matrix %d:\n", i);
        // for (int j = 54989849 - 5; j < 54989849 + 5; j++) {
        //     printf("%d ", h_prefix[i][j]);
        // }
        // printf("\n");
        // for (int j = 160666904 - 5; j < 160666904 + 5; j++) {
        //     printf("%d ", h_prefix[i][j]);
        // }
        // printf("\n");

        // check if prefix sum array is correct for frequency array
        // bool flag = true;
        // for (int j = 0; j < ranges[i] + 2; j++) {
        //     if (h_prefix[i][j+1] - h_prefix[i][j] != h_freq[i][j]) {
        //         cout << "went wront at : " << j <<  " " << h_freq[i][j] << " " << h_prefix[i][j+1] << " " << h_prefix[i][j] << endl;
        //         flag = false;
        //         break;
        //     }
        // }
        // cout << "flag: " << flag << endl;
        // int cnt = 0;
        // for (int j = 0; j < ranges[i] + 1; j++) {
        //     if (h_prefix[i][j] > h_prefix[i][j+1]) {
        //         cnt++;
        //         cout << j << ", " << h_prefix[i][j] << ", " << h_prefix[i][j+1] << endl;
        //         if (cnt == 50) break;
        //     }
        // }

        modify_matrix<<<(rows * cols + 255) / 256, 256>>>(d_matrices[i], d_prefix_copy[i], d_output[i], rows * cols);
        hipDeviceSynchronize();
        // delete[] h_prefix[i];
        // delete[] h_freq[i];
    }

    vector<vector<vector<int>>> modified_matrices(N);
    for (int i = 0; i < N; i++) {
        int rows = matrices[i].size();
        int cols = matrices[i][0].size();
        vector<int> flat_matrix(rows * cols);

        hipMemcpy(flat_matrix.data(), d_output[i], rows * cols * sizeof(int), hipMemcpyDeviceToHost);

        modified_matrices[i].resize(rows, vector<int>(cols));
        for (int r = 0; r < rows; r++)
            for (int c = 0; c < cols; c++)
                modified_matrices[i][r][c] = flat_matrix[r * cols + c];

        hipFree(d_matrices[i]);
        hipFree(d_freqs[i]);
        hipFree(d_prefix_copy[i]);
        hipFree(d_output[i]);
    }

    return modified_matrices;
}